#include "hip/hip_runtime.h"
/*
 * @file vector_2d_test.cu
 * @brief cuda test for Vector<Scalar, 2>.
 * @author Wei Chen
 * 
 * This file is part of Physika, a versatile physics simulation library.
 * Copyright (C) 2013- Physika Group.
 *
 * This Source Code Form is subject to the terms of the GNU General Public License v2.0. 
 * If a copy of the GPL was not distributed with this file, you can obtain one at:
 * http://www.gnu.org/licenses/gpl-2.0.html
 *
 */

#include "hip/hip_runtime.h"

#include "Physika_Core/Vectors/vector_2d.h"
#include "Physika_Core/Matrices/matrix_2x2.h"
#include "vector_2d_test.h"

namespace Physika{

namespace vector_2d_test{

__device__ inline void print(Vector2f vec)
{
    printf("(%f, %f)\n", vec[0], vec[1]);
}

__device__ inline void print(Matrix2f mat)
{
    for (int i = 0; i < 2; ++i)
    {
        for (int j = 0; j < 2; ++j)
            printf("%f, ", mat(i, j));
        printf("\n");
    }
}

__device__ void test_ctor()
{
    printf("test_ctor:\n");

    Vector2f default_vec;
    print(default_vec);

    Vector2f one_vec(1.0);
    print(one_vec);

    Vector2f ascend_vec(1.0, 2.0);
    print(ascend_vec);

    printf("dims: %d\n", Vector2f::dims());
    printf("\n");
}

__device__ void test_operator_asscess()
{
    printf("test_operator_asscess:\n");

    Vector2f ascend_vec(1.0, 2.0);
    printf("(%f, %f)\n", ascend_vec[0], ascend_vec[1]);
    printf("\n");
}

__device__ void test_operator_add()
{
    printf("test_operator_add:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    print(one_vec + two_vec);
    print(one_vec += two_vec);
    printf("\n");
}

__device__ void test_operator_minus()
{
    printf("test_operator_minus:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    print(one_vec - two_vec);
    print(one_vec -= two_vec);
    printf("\n");
}

__device__ void test_operator_assign()
{
    printf("test_operator_assign:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    print(one_vec = two_vec);
    printf("\n");
}

__device__ void test_operator_equal_not_equal()
{
    printf("test_operator_equal_not_equal:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);
    printf("one_vec == one_vec: %d\n", one_vec == one_vec);
    printf("one_vec == two_vec: %d\n", one_vec == two_vec);
    printf("one_vec != two_vec: %d\n", one_vec != two_vec);
    printf("\n");
}

__device__ void test_operator_add_scalar()
{
    printf("test_operator_add_scalar:\n");

    Vector2f one_vec(1.0);
    print(one_vec + 1.0f);
    print(one_vec += 1.0f);
    printf("\n");
}

__device__ void test_operator_minus_scalar()
{
    printf("test_operator_minus_scalar:\n");

    Vector2f one_vec(1.0);
    print(one_vec - 1.0f);
    print(one_vec -= 1.0f);
    printf("\n");
}

__device__ void test_operator_multi_scalar()
{
    printf("test_operator_multi_scalar:\n");

    Vector2f one_vec(1.0);
    print(one_vec * 2.0f);
    print(2.0f * one_vec);
    print(one_vec *= 2.0f);
    printf("\n");
}

__device__ void test_operator_sub_scalar()
{
    printf("test_operator_sub_scalar:\n");

    Vector2f one_vec(1.0);
    print(one_vec / 2.0f);
    print(one_vec /= 2.0f);
    printf("\n");
}

__device__ void test_operator_pre_minus()
{
    printf("test_operator_pre_minus:\n");

    Vector2f one_vec(1.0);
    print(-one_vec);
    printf("\n");
}

__device__ void test_norm()
{
    printf("test_norm:\n");

    Vector2f one_vec(1.0);
    printf("one_vec_norm: %f\n", one_vec.norm());

    Vector2f two_vec(2.0);
    printf("two_vec_norm: %f\n", two_vec.norm());

    printf("\n");
}

__device__ void test_norm_squared()
{
    printf("test_norm_squared:\n");

    Vector2f one_vec(1.0);
    printf("one_vec_normsquared: %f\n", one_vec.normSquared());

    Vector2f two_vec(2.0);
    printf("two_vec_normsquared: %f\n", two_vec.normSquared());

    printf("\n");
}

__device__ void test_normalize()
{
    printf("test_normalize:\n");

    Vector2f one_vec(1.0);
    print(one_vec.normalize());

    Vector2f two_vec(2.0);
    print(two_vec.normalize());

    printf("\n");
}

__device__ void test_cross()
{
    printf("test_cross:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    printf("one_vec.cross(two_vec): %f\n", one_vec.cross(two_vec));

    printf("\n");
}

__device__ void test_dot()
{
    printf("test_dot:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    printf("one_vec.dot(two_vec): %f\n", one_vec.dot(two_vec));

    printf("\n");
}

__device__ void test_outer_product()
{
    printf("test_outer_product:\n");

    Vector2f one_vec(1.0);
    Vector2f two_vec(2.0);

    printf("one_vec.outerProduct(two_vec):\n");
    print(one_vec.outerProduct(two_vec));

    printf("\n");
}

__global__ void test_vector_2d()
{
    test_ctor();
    test_operator_asscess();
    test_operator_add();
    test_operator_minus();
    test_operator_assign();
    test_operator_equal_not_equal();
    test_operator_add_scalar();
    test_operator_minus_scalar();
    test_operator_multi_scalar();
    test_operator_sub_scalar();
    test_operator_pre_minus();

    test_norm();
    test_norm_squared();
    test_normalize();
    test_cross();
    test_dot();
    test_outer_product();
}

}//end of namespace vector_2d_test

void testVector2d()
{
    vector_2d_test::test_vector_2d << <1, 1 >> > ();
    hipDeviceSynchronize();
}

}//end of namespace Physika