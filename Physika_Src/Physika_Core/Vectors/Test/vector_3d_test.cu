#include "hip/hip_runtime.h"
/*
 * @file vector_3d_test.cu
 * @brief cuda test for Vector<Scalar, 3>.
 * @author Wei Chen
 * 
 * This file is part of Physika, a versatile physics simulation library.
 * Copyright (C) 2013- Physika Group.
 *
 * This Source Code Form is subject to the terms of the GNU General Public License v2.0. 
 * If a copy of the GPL was not distributed with this file, you can obtain one at:
 * http://www.gnu.org/licenses/gpl-2.0.html
 *
 */

#include "hip/hip_runtime.h"

#include "Physika_Core/Vectors/vector_3d.h"
#include "Physika_Core/Matrices/matrix_3x3.h"
#include "vector_3d_test.h"

namespace Physika{

namespace vector_3d_test{

__device__ inline void print(Vector3f vec)
{
    printf("(%f, %f, %f)\n", vec[0], vec[1], vec[2]);
}

__device__ inline void print(Matrix3f mat)
{
    for (int i = 0; i < 3; ++i)
    {
        for (int j = 0; j < 3; ++j)
            printf("%f, ", mat(i, j));
        printf("\n");
    }
}

__device__ void test_ctor()
{
    printf("test_ctor:\n");

    Vector3f default_vec;
    print(default_vec);

    Vector3f one_vec(1.0);
    print(one_vec);

    Vector3f ascend_vec(1.0, 2.0, 3.0);
    print(ascend_vec);

    printf("dims: %d\n", Vector3f::dims());
    printf("\n");
}

__device__ void test_operator_asscess()
{
    printf("test_operator_asscess:\n");

    Vector3f ascend_vec(1.0, 2.0, 3.0);
    printf("(%f, %f, %f)\n", ascend_vec[0], ascend_vec[1], ascend_vec[2]);
    printf("\n");
}

__device__ void test_operator_add()
{
    printf("test_operator_add:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    print(one_vec + two_vec);
    print(one_vec += two_vec);
    printf("\n");
}

__device__ void test_operator_minus()
{
    printf("test_operator_minus:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    print(one_vec - two_vec);
    print(one_vec -= two_vec);
    printf("\n");
}

__device__ void test_operator_assign()
{
    printf("test_operator_assign:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    print(one_vec = two_vec);
    printf("\n");
}

__device__ void test_operator_equal_not_equal()
{
    printf("test_operator_equal_not_equal:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);
    printf("one_vec == one_vec: %d\n", one_vec == one_vec);
    printf("one_vec == two_vec: %d\n", one_vec == two_vec);
    printf("one_vec != two_vec: %d\n", one_vec != two_vec);
    printf("\n");
}

__device__ void test_operator_add_scalar()
{
    printf("test_operator_add_scalar:\n");

    Vector3f one_vec(1.0);
    print(one_vec + 1.0f);
    print(one_vec += 1.0f);
    printf("\n");
}

__device__ void test_operator_minus_scalar()
{
    printf("test_operator_minus_scalar:\n");

    Vector3f one_vec(1.0);
    print(one_vec - 1.0f);
    print(one_vec -= 1.0f);
    printf("\n");
}

__device__ void test_operator_multi_scalar()
{
    printf("test_operator_multi_scalar:\n");

    Vector3f one_vec(1.0);
    print(one_vec * 2.0f);
    print(2.0f * one_vec);
    print(one_vec *= 2.0f);
    printf("\n");
}

__device__ void test_operator_sub_scalar()
{
    printf("test_operator_sub_scalar:\n");

    Vector3f one_vec(1.0);
    print(one_vec / 2.0f);
    print(one_vec /= 2.0f);
    printf("\n");
}

__device__ void test_operator_pre_minus()
{
    printf("test_operator_pre_minus:\n");

    Vector3f one_vec(1.0);
    print(-one_vec);
    printf("\n");
}

__device__ void test_norm()
{
    printf("test_norm:\n");

    Vector3f one_vec(1.0);
    printf("one_vec_norm: %f\n", one_vec.norm());

    Vector3f two_vec(2.0);
    printf("two_vec_norm: %f\n", two_vec.norm());

    printf("\n");
}

__device__ void test_norm_squared()
{
    printf("test_norm_squared:\n");

    Vector3f one_vec(1.0);
    printf("one_vec_normsquared: %f\n", one_vec.normSquared());

    Vector3f two_vec(2.0);
    printf("two_vec_normsquared: %f\n", two_vec.normSquared());

    printf("\n");
}

__device__ void test_normalize()
{
    printf("test_normalize:\n");

    Vector3f one_vec(1.0);
    print(one_vec.normalize());

    Vector3f two_vec(2.0);
    print(two_vec.normalize());

    printf("\n");
}

__device__ void test_cross()
{
    printf("test_cross:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    printf("one_vec.cross(two_vec):\n");
    print(one_vec.cross(two_vec));

    printf("\n");
}

__device__ void test_dot()
{
    printf("test_dot:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    printf("one_vec.dot(two_vec): %f\n", one_vec.dot(two_vec));

    printf("\n");
}

__device__ void test_outer_product()
{
    printf("test_outer_product:\n");

    Vector3f one_vec(1.0);
    Vector3f two_vec(2.0);

    printf("one_vec.outerProduct(two_vec):\n");
    print(one_vec.outerProduct(two_vec));

    printf("\n");
}

__global__ void test_vector_3d()
{
    test_ctor();
    test_operator_asscess();
    test_operator_add();
    test_operator_minus();
    test_operator_assign();
    test_operator_equal_not_equal();
    test_operator_add_scalar();
    test_operator_minus_scalar();
    test_operator_multi_scalar();
    test_operator_sub_scalar();
    test_operator_pre_minus();

    test_norm();
    test_norm_squared();
    test_normalize();
    test_cross();
    test_dot();
    test_outer_product();
}

}//end of namespace vector_3d_test

void testVector3d()
{
    vector_3d_test::test_vector_3d << <1, 1 >> > ();
    hipDeviceSynchronize();
}

}//end of namespace Physika