#include "hip/hip_runtime.h"
/*
 * @file vector_4d_test.cu
 * @brief cuda test for Vector<Scalar, 4>.
 * @author Wei Chen
 * 
 * This file is part of Physika, a versatile physics simulation library.
 * Copyright (C) 2013- Physika Group.
 *
 * This Source Code Form is subject to the terms of the GNU General Public License v2.0. 
 * If a copy of the GPL was not distributed with this file, you can obtain one at:
 * http://www.gnu.org/licenses/gpl-2.0.html
 *
 */

#include "hip/hip_runtime.h"

#include "Physika_Core/Vectors/vector_4d.h"
#include "Physika_Core/Matrices/matrix_4x4.h"
#include "vector_4d_test.h"

namespace Physika{

namespace vector_4d_test{

__device__ inline void print(Vector4f vec)
{
    printf("(%f, %f, %f, %f)\n", vec[0], vec[1], vec[2], vec[3]);
}

__device__ inline void print(Matrix4f mat)
{
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
            printf("%f, ", mat(i, j));
        printf("\n");
    }
}

__device__ void test_ctor()
{
    printf("test_ctor:\n");

    Vector4f default_vec;
    print(default_vec);

    Vector4f one_vec(1.0);
    print(one_vec);

    Vector4f ascend_vec(1.0, 2.0, 3.0, 4.0);
    print(ascend_vec);

    printf("dims: %d\n", Vector4f::dims());
    printf("\n");
}

__device__ void test_operator_asscess()
{
    printf("test_operator_asscess:\n");

    Vector4f ascend_vec(1.0, 2.0, 3.0, 4.0);
    printf("(%f, %f, %f, %f)\n", ascend_vec[0], ascend_vec[1], ascend_vec[2], ascend_vec[3]);
    printf("\n");
}

__device__ void test_operator_add()
{
    printf("test_operator_add:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);

    print(one_vec + two_vec);
    print(one_vec += two_vec);
    printf("\n");
}

__device__ void test_operator_minus()
{
    printf("test_operator_minus:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);

    print(one_vec - two_vec);
    print(one_vec -= two_vec);
    printf("\n");
}

__device__ void test_operator_assign()
{
    printf("test_operator_assign:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);

    print(one_vec = two_vec);
    printf("\n");
}

__device__ void test_operator_equal_not_equal()
{
    printf("test_operator_equal_not_equal:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);
    printf("one_vec == one_vec: %d\n", one_vec == one_vec);
    printf("one_vec == two_vec: %d\n", one_vec == two_vec);
    printf("one_vec != two_vec: %d\n", one_vec != two_vec);
    printf("\n");
}

__device__ void test_operator_add_scalar()
{
    printf("test_operator_add_scalar:\n");

    Vector4f one_vec(1.0);
    print(one_vec + 1.0f);
    print(one_vec += 1.0f);
    printf("\n");
}

__device__ void test_operator_minus_scalar()
{
    printf("test_operator_minus_scalar:\n");

    Vector4f one_vec(1.0);
    print(one_vec - 1.0f);
    print(one_vec -= 1.0f);
    printf("\n");
}

__device__ void test_operator_multi_scalar()
{
    printf("test_operator_multi_scalar:\n");

    Vector4f one_vec(1.0);
    print(one_vec * 2.0f);
    print(2.0f * one_vec);
    print(one_vec *= 2.0f);
    printf("\n");
}

__device__ void test_operator_sub_scalar()
{
    printf("test_operator_sub_scalar:\n");

    Vector4f one_vec(1.0);
    print(one_vec / 2.0f);
    print(one_vec /= 2.0f);
    printf("\n");
}

__device__ void test_operator_pre_minus()
{
    printf("test_operator_pre_minus:\n");

    Vector4f one_vec(1.0);
    print(-one_vec);
    printf("\n");
}

__device__ void test_norm()
{
    printf("test_norm:\n");

    Vector4f one_vec(1.0);
    printf("one_vec_norm: %f\n", one_vec.norm());

    Vector4f two_vec(2.0);
    printf("two_vec_norm: %f\n", two_vec.norm());

    printf("\n");
}

__device__ void test_norm_squared()
{
    printf("test_norm_squared:\n");

    Vector4f one_vec(1.0);
    printf("one_vec_normsquared: %f\n", one_vec.normSquared());

    Vector4f two_vec(2.0);
    printf("two_vec_normsquared: %f\n", two_vec.normSquared());

    printf("\n");
}

__device__ void test_normalize()
{
    printf("test_normalize:\n");

    Vector4f one_vec(1.0);
    print(one_vec.normalize());

    Vector4f two_vec(2.0);
    print(two_vec.normalize());

    printf("\n");
}

__device__ void test_dot()
{
    printf("test_dot:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);

    printf("one_vec.dot(two_vec): %f\n", one_vec.dot(two_vec));

    printf("\n");
}

__device__ void test_outer_product()
{
    printf("test_outer_product:\n");

    Vector4f one_vec(1.0);
    Vector4f two_vec(2.0);

    printf("one_vec.outerProduct(two_vec):\n");
    print(one_vec.outerProduct(two_vec));

    printf("\n");
}

__global__ void test_vector_4d()
{
    test_ctor();
    test_operator_asscess();
    test_operator_add();
    test_operator_minus();
    test_operator_assign();
    test_operator_equal_not_equal();
    test_operator_add_scalar();
    test_operator_minus_scalar();
    test_operator_multi_scalar();
    test_operator_sub_scalar();
    test_operator_pre_minus();

    test_norm();
    test_norm_squared();
    test_normalize();
    test_dot();
    test_outer_product();
}

}//end of namespace vector_4d_test

void testVector4d()
{
    vector_4d_test::test_vector_4d << <1, 1 >> > ();
    hipDeviceSynchronize();
}

}//end of namespace Physika