#include "hip/hip_runtime.h"
#include "Function1Pt.h"
#include "Physika_Core/Utilities/cuda_helper_math.h"
#include "Physika_Core/Utilities/cuda_utilities.h"
namespace Physika
{
	namespace Function1Pt
	{
		template<typename T1, typename T2>
		__global__ void KerLength(T1* lhs, T2* rhs, int num)
		{
			int pId = threadIdx.x + (blockIdx.x * blockDim.x);
			if (pId >= num) return;

			lhs[pId] = length(rhs[pId]);
		}

		template<typename T1, typename T2>
		void Length(DeviceArray<T1>& lhs, DeviceArray<T2>& rhs)
		{
			assert(lhs.size() == rhs.size());
			unsigned pDim = cudaGridSize(rhs.size(), BLOCK_SIZE);
			KerLength << <pDim, BLOCK_SIZE >> > (lhs.getDataPtr(), rhs.getDataPtr(), lhs.size());
		}
	}
}