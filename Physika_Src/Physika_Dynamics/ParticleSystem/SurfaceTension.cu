#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Physika_Core/Utilities/cuda_utilities.h"
#include "Physika_Core/Utilities/template_functions.h"
#include "SurfaceTension.h"


namespace Physika
{
// 	struct ST_STATE
// 	{
// 		float mass;
// 		float smoothingLength;
// 		float restDensity;
// 		SmoothKernel kernSmooth;
// 	};

//	__constant__ ST_STATE const_st_state;


	template<typename Real, typename Coord>
	__global__ void ST_ComputeSurfaceEnergy
	(
		DeviceArray<Real> energyArr,
		DeviceArray<Coord> posArr,
		DeviceArray<NeighborList> neighbors,
		Real smoothingLength
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.Size()) return;

		Real total_weight = Real(0);
		Coord dir_i = Make<Coord>(0);

		SmoothKernel<Real> kern;

		Coord pos_i = posArr[pId];
		int nbSize = neighbors[pId].size;
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors[pId][ne];
			Real r = length(pos_i - posArr[j]);

			if (r > EPSILON)
			{
				Real weight = -kern.Gradient(r, smoothingLength);
				total_weight += weight;
				dir_i += (posArr[j] - pos_i)*(weight / r);
			}
		}

		total_weight = total_weight < EPSILON ? 1.0f : total_weight;
		Real absDir = length(dir_i) / total_weight;

		energyArr[pId] = absDir*absDir;
	}

	template<typename Real, typename Coord>
	__global__ void ST_ComputeSurfaceTension
	(
		DeviceArray<Coord> velArr, 
		DeviceArray<Real> energyArr, 
		DeviceArray<Coord> posArr, 
		DeviceArray<Attribute> attArr,
		DeviceArray<NeighborList> neighbors, 
		Real smoothingLength,
		Real mass,
		Real restDensity,
		float dt
	)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.Size()) return;
		if (!attArr[pId].IsDynamic()) return;

		Real Vref = mass / restDensity;


		float alpha = (float) 945.0f / (32.0f * (float)M_PI * smoothingLength * smoothingLength * smoothingLength);
		float ceof = 16000.0f * alpha;

		SmoothKernel<Real> kern;

		Coord F_i = Make<Coord>(0);
		Coord dv_pi = Make<Coord>(0);
		Coord pos_i = posArr[pId];
		int nbSize = neighbors[pId].size;
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = neighbors[pId][ne];
			float r = length(pos_i - posArr[j]);

			if (r > EPSILON)
			{
				Coord temp = Vref*Vref*kern.Gradient(r, smoothingLength)*(posArr[j] - pos_i) * (1.0f / r);
				Coord dv_ij = dt * ceof*1.0f*(energyArr[pId])*temp / mass;
				F_i += dv_ij;

// 				atomicAdd(&velArr[j].x, dv_ij.x);
// 				atomicAdd(&velArr[j].y, dv_ij.y);
// 				atomicAdd(&velArr[j].z, dv_ij.z);
			}
		}
		velArr[pId] -= F_i;

// 		atomicAdd(&velArr[pId].x, -F_i.x);
// 		atomicAdd(&velArr[pId].y, -F_i.y);
// 		atomicAdd(&velArr[pId].z, -F_i.z);
	}

	template<typename TDataType>
	SurfaceTension<TDataType>::SurfaceTension(ParticleSystem<TDataType>* parent)
		:Module()
		,m_parent(parent)
	{
		assert(m_parent != NULL);

		setInputSize(2);
		setOutputSize(1);

		int num = m_parent->GetParticleNumber();

		m_energy = DeviceBuffer<Real>::create(num);

		updateStates();
	}

	template<typename TDataType>
	bool SurfaceTension<TDataType>::execute()
	{
		DeviceArray<Coord>* posArr = m_parent->GetNewPositionBuffer()->getDataPtr();
		DeviceArray<Coord>* velArr = m_parent->GetNewVelocityBuffer()->getDataPtr();
		DeviceArray<Attribute>* attArr = m_parent->GetAttributeBuffer()->getDataPtr();
		float dt = m_parent->getDt();

		DeviceArray<NeighborList>* neighborArr = m_parent->GetNeighborBuffer()->getDataPtr();

		DeviceArray<Real>* energy = m_energy->getDataPtr();

		Real mass = m_parent->GetParticleMass();
		Real smoothingLength = m_parent->GetSmoothingLength();
		Real restDensity = m_parent->GetRestDensity();

		uint pDims = cudaGridSize(posArr->Size(), BLOCK_SIZE);
		ST_ComputeSurfaceEnergy <Real, Coord> << < pDims, BLOCK_SIZE >> > (*energy, *posArr, *neighborArr, smoothingLength);
		ST_ComputeSurfaceTension <Real, Coord> << < pDims, BLOCK_SIZE >> > (*velArr, *energy, *posArr, *attArr, *neighborArr, smoothingLength, mass, restDensity, dt);

		return true;
	}

	template<typename TDataType>
	bool SurfaceTension<TDataType>::updateStates()
	{
// 		ST_STATE cm;
// 		cm.mass = m_parent->GetParticleMass();
// 		cm.smoothingLength = m_parent->GetSmoothingLength();
// 		cm.kernSmooth = SmoothKernel();
// 		cm.restDensity = m_parent->GetRestDensity();
// 
// 		hipMemcpyToSymbol(HIP_SYMBOL(const_st_state), &cm, sizeof(ST_STATE));

		return true;
	}

}