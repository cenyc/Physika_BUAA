#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Physika_Core/Utilities/cuda_utilities.h"
#include "Physika_Core/Utilities/template_functions.h"
#include "ParticlePrediction.h"

namespace Physika
{
// 	struct PP_STATE
// 	{
// 		float3 bodyForce;
// 	};
// 
// 	__constant__ PP_STATE const_pp_state;

	template<typename Coord>
	__global__ void PP_Predict(
		DeviceArray<Coord> posArr,
		DeviceArray<Coord> velArr,
		Coord bodyForce,
		float dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.Size()) return;

		Coord pos_i = posArr[pId];
		Coord vel_i = velArr[pId];

		pos_i += vel_i*dt;
		vel_i += bodyForce*dt;
//		vel_i += 20.0f*(make_float3(0.5f, 0.2f, 0.5f) - posArr[pId])*dt;

		posArr[pId] = pos_i;
		velArr[pId] = vel_i;
	}

	template<typename Coord>
	__global__ void PP_PredictPosition(
		DeviceArray<Coord> posArr,
		DeviceArray<Coord> velArr,
		DeviceArray<Attribute> attriArr, 
		float dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.Size()) return;

		if (!attriArr[pId].IsFixed())
		{
			Coord pos_i = posArr[pId];
			Coord vel_i = velArr[pId];

			pos_i += vel_i*dt;

//			vel_i += 20.0f*(make_float3(0.5f) - posArr[pId])*dt;

#ifdef SIMULATION2D
			pos_i.z = 0.5f;
#endif
			posArr[pId] = pos_i;
			velArr[pId] = vel_i;

// 			if (attriArr[pId].IsPassive() && pos_i.y < 0.85f)
// 			{
// 				attriArr[pId].SetDynamic();
// 			}
		}
	}

	template<typename Coord>
	__global__ void PP_PredictVelocity(
		DeviceArray<Coord> velArr,
		DeviceArray<Attribute> attriArr, 
		Coord bodyForce,
		float dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velArr.Size()) return;

		if (attriArr[pId].IsDynamic())
		{
			Coord vel_i = velArr[pId];
			vel_i += bodyForce*dt;

			velArr[pId] = vel_i;
		}
	}

	template<typename Coord>
	__global__ void PP_CorrectPosition(
		DeviceArray<Coord> newPos, 
		DeviceArray<Coord> oldPos,
		DeviceArray<Coord> newVel,
		DeviceArray<Coord> oldVel,
		DeviceArray<Attribute> attriArr, 
		float dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= newPos.Size()) return;

		if (!attriArr[pId].IsFixed())
		{
			newPos[pId] = oldPos[pId] + 0.5f * dt * (oldVel[pId] + newVel[pId]);

#ifdef SIMULATION2D
			newPos[pId].z = 0.5f;
#endif
		}
	}

	template<typename TDataType>
	ParticlePrediction<TDataType>::ParticlePrediction(ParticleSystem<TDataType>* parent)
		:Module()
		,m_parent(parent)
	{
		assert(m_parent != NULL);

		setInputSize(1);
		setOutputSize(1);

		updateStates();
	}

	template<typename TDataType>
	bool ParticlePrediction<TDataType>::execute()
	{
		DeviceArray<Coord>* posArr = m_parent->GetNewPositionBuffer()->getDataPtr();
		DeviceArray<Coord>* velArr = m_parent->GetNewVelocityBuffer()->getDataPtr();
		DeviceArray<Attribute>* attriArr = m_parent->GetAttributeBuffer()->getDataPtr();
		Coord gravity = m_parent->GetGravity();
		float dt = m_parent->getDt();
		
		uint pDims = cudaGridSize(posArr->Size(), BLOCK_SIZE);
		
		PP_Predict <Coord> << <pDims, BLOCK_SIZE >> > (*posArr, *velArr, gravity, dt);

		return true;
	}

	template<typename TDataType>
	void ParticlePrediction<TDataType>::PredictPosition(float dt)
	{
		DeviceArray<Coord>* posArr = m_parent->GetNewPositionBuffer()->getDataPtr();
		DeviceArray<Coord>* velArr = m_parent->GetNewVelocityBuffer()->getDataPtr();
		DeviceArray<Attribute>* attriArr = m_parent->GetAttributeBuffer()->getDataPtr();

		uint pDims = cudaGridSize(posArr->Size(), BLOCK_SIZE);
		PP_PredictPosition <Coord> << <pDims, BLOCK_SIZE >> > (*posArr, *velArr, *attriArr, dt);
	}

	template<typename TDataType>
	void ParticlePrediction<TDataType>::PredictVelocity(float dt)
	{
		DeviceArray<Coord>* velArr = m_parent->GetNewVelocityBuffer()->getDataPtr();
		DeviceArray<Attribute>* attriArr = m_parent->GetAttributeBuffer()->getDataPtr();

		uint pDims = cudaGridSize(velArr->Size(), BLOCK_SIZE);
		Coord gravity = Make<Coord>(0);
		PP_PredictVelocity <Coord> << <pDims, BLOCK_SIZE >> > (*velArr, *attriArr, gravity, dt);
	}

	template<typename TDataType>
	void ParticlePrediction<TDataType>::CorrectPosition(float dt)
	{
		DeviceArray<Coord>* oldPos = m_parent->GetOldPositionBuffer()->getDataPtr();
		DeviceArray<Coord>* newPos = m_parent->GetNewPositionBuffer()->getDataPtr();
		DeviceArray<Coord>* oldVel = m_parent->GetOldVelocityBuffer()->getDataPtr();
		DeviceArray<Coord>* newVel = m_parent->GetNewVelocityBuffer()->getDataPtr();
		DeviceArray<Attribute>* attriArr = m_parent->GetAttributeBuffer()->getDataPtr();

		uint pDims = cudaGridSize(oldPos->Size(), BLOCK_SIZE);
		PP_CorrectPosition << <pDims, BLOCK_SIZE >> > (*newPos, *oldPos, *newVel, *oldVel, *attriArr, dt);
	}

	template<typename TDataType>
	bool ParticlePrediction<TDataType>::updateStates()
	{
// 		PP_STATE cm;
// 		cm.bodyForce = m_parent->GetBodyForce();
// 
// 		hipMemcpyToSymbol(HIP_SYMBOL(const_pp_state), &cm, sizeof(PP_STATE));

		return true;
	}

}